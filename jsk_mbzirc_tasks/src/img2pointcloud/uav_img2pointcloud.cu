#include "hip/hip_runtime.h"
/*
 jsk_mbzirc_task
 cuda accelerate
 */

// Author: Chen

//opencv
#include <cv_bridge/cv_bridge.h>

//cuda
#include <include/hip/hip_runtime.h>
#include <include/hip/hip_runtime.h>
//pcl
#include <stdlib.h>
#include <pcl/point_cloud.h>
#include <pcl/point_types.h>
#include <iostream>
#define Ground_Z 0.0
typedef pcl::PointXYZRGB PointTYPE;


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void cuda_projecting(double *a, double *b, double *c, uchar *imgdata, float *pcdata)
{
    float A[2][2],bv[2];
//    int i = blockIdx.x; //row
//    int j = blockIdx.y; //column
//    int offset = j + i*gridDim.y;
    int i = blockIdx.x;
    int j = threadIdx.x;
    int offset = j + i*blockDim.x;
    A[0][0] = j*c[0] - a[0]; A[0][1] = j*c[1] - a[1];
    A[1][0] = i*c[0] - b[0]; A[1][1] = i*c[1] - b[1];
    bv[0]= a[2]*Ground_Z + a[3] - j*c[2]*Ground_Z - j*c[3];
    bv[1] = b[2]*Ground_Z + b[3] - i*c[2]*Ground_Z - i*c[3];
    float DomA = A[1][1]*A[0][0]-A[0][1]*A[1][0];
    int offsetimg = offset*3;  //3 channel
    int offsetpc = offset*( sizeof(PointTYPE)/sizeof(float));
    pcdata[offsetpc] = (A[1][1]*bv[0]-A[0][1]*bv[1])/DomA;
    pcdata[offsetpc+1] = (A[0][0]*bv[1]-A[1][0]*bv[0])/DomA;
    pcdata[offsetpc+2] = (float)Ground_Z;
    uint8_t rgb[4];
    rgb[0] = imgdata[offsetimg]; rgb[1] = imgdata[offsetimg+1];
    rgb[2] = imgdata[offsetimg+2]; rgb[3] = 0;
    pcdata[offsetpc+3] = 1.0;
    pcdata[offsetpc+4] = *(float *)(rgb);
}

float process_in_cuda(double *_a, double *_b,double *_c,
                    cv::Mat *_img, pcl::PointCloud<PointTYPE> *PC)
{
    uchar *dev_imgdata;
    float *dev_pcdata;
    const int coefsize = 4*sizeof(double);
    const int pixelsize = _img->rows*_img->cols;
    const int pointcloudsize = pixelsize*sizeof(PointTYPE);
    double *dev_a, *dev_b, *dev_c;

    //malloc the mem
    hipSetDevice(0);
    HANDLE_ERROR(hipMalloc((void **)&dev_a, coefsize));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, coefsize));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, coefsize));
    HANDLE_ERROR(hipMalloc((void **)&dev_imgdata,sizeof(uchar)*pixelsize*_img->channels()));
    HANDLE_ERROR(hipMalloc((void **)&dev_pcdata, pixelsize*sizeof(PointTYPE)));

    //copy to device mem
    HANDLE_ERROR(hipMemcpy(dev_a,_a,coefsize,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b,_b,coefsize,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_c,_c,coefsize,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_imgdata,_img->data,
                            sizeof(uchar)*pixelsize*_img->channels(),hipMemcpyHostToDevice));

    //create cudaevent to record the time...
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //run device function...
//    dim3 grid(PC->height,PC->width);
//    cuda_projecting<<<grid,1>>>(dev_a, dev_b, dev_c, dev_imgdata, dev_pcdata);
    cuda_projecting<<<PC->height,PC->width>>>(dev_a, dev_b, dev_c, dev_imgdata, dev_pcdata);
    //copy back to pointcloud...
    HANDLE_ERROR(hipMemcpy(PC->points.data(),dev_pcdata,pointcloudsize,hipMemcpyDeviceToHost));
    hipEventRecord( stop, 0 );

    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    hipFree(dev_imgdata); hipFree(dev_pcdata);

   return elapsedTime;

}
